#include "hip/hip_runtime.h"
/* calc_cuda.cu -- CUDA engine, which is only included if the platform supports
                -- it

  This file is part of the fractalexplorer project.

  fractalexplorer source code, as well as any other resources in this
project are free software; you are free to redistribute it and/or modify them
under the terms of the GNU General Public License; either version 3 of the
license, or any later version.

  These programs are hopefully useful and reliable, but it is understood
that these are provided WITHOUT ANY WARRANTY, or MERCHANTABILITY or FITNESS
FOR A PARTICULAR PURPOSE. See the GPLv3 or email at
<cade@cade.site> for more info on this.

  Here is a copy of the GPL v3, which this software is licensed under. You
can also find a copy at http://www.gnu.org/licenses/.

*/

#include <hip/hip_complex.h>


extern "C" {

#include <stdio.h>
#include <stdlib.h>

#include "fr.h"
#include "log.h"


/*

our complex number library, in cuda

*/


// behave just like the C functions
#define creal(a) a.x
#define cimag(a) a.y
#define cabs(a) sqrt(a.x*a.x+a.y*a.y)
#define carg(a) atan2(a.y, a.x)


// the squared abs, i.e. cabs(a) * cabs(a)
#define cabs2(a) (a.x*a.x+a.y*a.y)



// constructor
#define ccreate(x, y) ((hipDoubleComplex){ (x), (y) })


// pow, returns e**(x)
__host__ __device__ static __inline__
hipDoubleComplex cuCexp(hipDoubleComplex x) {
    hipDoubleComplex result = { 0.0, 0.0 };
    double tmp_scale = exp(x.x);
    sincos(x.y, &result.y, &result.x);
    result.x *= tmp_scale;
    result.y *= tmp_scale;
    return result;
}

// 1 / x
__host__ __device__ static __inline__
hipDoubleComplex cuCrec(hipDoubleComplex x) {
    double to_scale = cabs2(x);
    return ccreate(x.x / to_scale, -x.y / to_scale);
}

// natural logarithm, base e ~= 2.71828
__host__ __device__ static __inline__
hipDoubleComplex cuClog(hipDoubleComplex x) {
    return ccreate(log(cabs2(x)) / 2.0, atan2(x.y, x.x));
}

// log_y(x), or log base y of x
__host__ __device__ static __inline__
hipDoubleComplex cuClogb(hipDoubleComplex x, hipDoubleComplex y) {
    return hipCdiv(cuClog(x), cuClog(y));
}

// x * x, or x ** 2, x squared
__host__ __device__ static __inline__
hipDoubleComplex cuCsqr(hipDoubleComplex x) {
    return ccreate(x.x*x.x - x.y*x.y, 2 * x.x*x.y);
}

// x ** y, but optimized for integers
__host__ __device__ static __inline__
hipDoubleComplex cuCpowi(hipDoubleComplex x, int y) {
    bool is_neg_pow = y < 0;
    y = abs(y);
    // holds track of x^{2^{exp bit}}
    hipDoubleComplex xt2eb = x, result = ccreate(1, 0);
    while (y > 0) {
        if (y & 1) {
            result = hipCmul(result, xt2eb);
        }
        xt2eb = cuCsqr(xt2eb);
        y >>= 1;
    }
    return (is_neg_pow) ? cuCrec(result) : result;
}


// x ** y, or x to the y power
__host__ __device__ static __inline__
hipDoubleComplex cuCpow(hipDoubleComplex x, hipDoubleComplex y) {
    return cuCexp(hipCmul(cuClog(x), y));
}

// sin, returns sin(x). Highly optimized method
// correct for all complex numbers
__host__ __device__ static __inline__
hipDoubleComplex cuCsin(hipDoubleComplex x) {
    return ccreate(sin(x.x) * cosh(x.y), cos(x.x) * sinh(x.y));
}


// cos, returns cos(x). Highly optimized and
// works for complex numbers
__host__ __device__ static __inline__
hipDoubleComplex cuCcos(hipDoubleComplex x) {
    return ccreate(cos(x.x) * cosh(x.y), -sin(x.x) * sinh(x.y));
}


// a macro to check a result and then print out info if failed, and (possibly)
// exit
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line) {
   if (code != hipSuccess) {
      log_error("GPUassert (at %s:%d) (code %d): %s\n", file, line, code,
                      hipGetErrorString(code));

      // TODO: determine when to exit. Some kernel launch failures seem to be
      // recoverable
      // codes: 35 is insufficient driver
      if (code != 35) {
          exit(code);
      }
   }
}

// last width and height
int lw = 0, lh = 0;

// number of colors
int colnum;

// GPU buffers
unsigned char * _gpu_output = NULL, * _gpu_col = NULL;
int * _gpu_err = NULL;
bool cuda_has_init = false;


// a CUDA device kernel to compute fractal pixels value. Takes fr as parameters,
// my_h and my_off for custom rank based parameters, color and the number of
// colors, and an output buffer. err will be set to non-zero if an error occured
// Note that the buffers should be allocated with CUDA device memory functions
__global__
void cuda_kernel(fr_t fr, int tid, int threads, unsigned char * col, int ncol, unsigned char * output, int * err) {

    // compute the current pixel offset
    int px = (blockIdx.x * blockDim.x) + threadIdx.x;
    int py = tid + threads * ((blockIdx.y * blockDim.y) + threadIdx.y);

    // it may be desirable to launch a job with more than neccessary dimensions
    // so, we simply don't run when this is true
    if (px >= fr.w || py >= fr.h) {
        return;
    }

    // the return index, current iteration, and color indexes
    int ri = 4 * (px + fr.w * (py / threads)), ci, c0, c1;


    // fractional index
    double fri, mfact, _q, tmp;
    
    // minimum magnitude, and the iteration it occured at (only used by some functions)
    double min_mag;
    int min_mag_ci;

    // c componenets, and temporary variables
    double c_r, c_i, _t0, _t1, _t2, _t3;

    // real, imaginary
    hipDoubleComplex z, c, q;
    
    c_r = fr.cX - (fr.w - 2 * px) / (fr.Z * fr.w);
    c_i = fr.cY + (fr.h - 2 * py) / (fr.Z * fr.w);
    
    c = ccreate(c_r, c_i);

    z = c;

    // u + i*v
    q = ccreate(fr.u, fr.v);

    switch (fr.fractal_type) {
        case FR_MANDELBROT:
            // see above in this file, this method determines whether
            // we should skip the computation
            _q = (z.x - .25f);
            _q = _q * _q + z.y * z.y;
            if (_q * (_q + (z.x - .25f)) < z.y * z.y / 4.0f) {
                ci = fr.max_iter;
                fri = ci + 0.0;
            } else {
                _t0 = z.x * z.x;
                _t1 = z.y * z.y;
                for (ci = 0; ci < fr.max_iter && _t0 + _t1 < 256.0; ++ci) {
                    _t2 = 2 * z.x * z.y;
                    z.x = _t0 - _t1 + c_r;
                    z.y = _t2 + c_i;
                    _t0 = z.x * z.x; _t1 = z.y * z.y;
                    //z = cuCsqr(z);
                    //z = hipCadd(z, c);
                }
                tmp = log(log(_t0 + _t1));
                fri = 2.0 + ci - tmp / log(2.0);
            }
            break;
        case FR_MANDELBROT_3:
            // similar to the default mandelbrot, we will loop and use
            // an escape value of 16.0. However, we have no speedups,
            // like bulb_check_0 for this, so we will just iterate
            // the function
            for (ci = 0; ci < fr.max_iter && cabs(z) <= 16.0; ++ci) {
                z = hipCmul(z, hipCmul(z, z));
                z = hipCadd(z, c);
            }


            // we use the same basic method, but with a different
            // divisor. The divisor to get even boundaries between
            // iteration bounds is log(3.0) (log of the exponent),
            // however try 2.5 or 3.5 or 5 to get other, cool
            // fractional iteration counts
            tmp = log(log(creal(z)*creal(z) + cimag(z)*cimag(z)))
                  / log(3.0);
            fri = 2.0 + ci - tmp;
            break;
        case FR_EXP:
            //
            for (ci = 0; ci < fr.max_iter && fabs(creal(z)) < 16.0; ++ci) {
                z = cuCexp(z);
                z = hipCadd(z, c);
            }
            // no current way to easily do a fractional iteration, so
            // just send a fractional iteration of the actual integer
            // value
            fri = 0.0 + ci;
            break;
        case FR_SIN:
            // the sin(z)+c may not just escape from a radius, and we
            // should check that the imaginary portion escapes
            for (ci = 0; ci < fr.max_iter && fabs(cimag(z)) < 16.0; ++ci) {
                z = cuCsin(z);
                z = hipCadd(z, c);

            }
            // no current way to easily do a fractional iteration, so
            // just send a fractional iteration of the actual integer
            // value
            fri = 0.0 + ci;
            break;
        case FR_JULIA:
            // z**2 + q
            for (ci = 0; ci < fr.max_iter && cabs(z) < 16.0; ++ci) {
                z = cuCsqr(z);
                z = hipCadd(z, q);

            }
            // no current way to easily do a fractional iteration, so
            // just send a fractional iteration of the actual integer
            // value
            fri = 2.0 + ci - log(log(cabs2(z))) / log(2.0);
            break;
        default:
            // this should never happen
            *err = 1;
            return;
            break;
    }

    // if ci is set to default values, if they have set ci to 0 or max
    // the computation might iter,
    // set the fri to corresponding values
    if (ci == fr.max_iter) {
        fri = 0.0 + fr.max_iter;
    }

    if (fr.fractal_flags & FRF_ADD_PERIOD) {
        tmp = fri - floor(fri);
        fri += ((1-tmp)*carg(z)+tmp*(carg(hipCadd(hipCmul(z, z), c))));
    }

    if (fr.fractal_flags & FRF_BINARYDECOMP_REAL && creal(z) >= 0) {
        fri += 1.0;
    }
    
    if (fr.fractal_flags & FRF_BINARYDECOMP_IMAG && cimag(z) >= 0) {
        fri += 2.0;
    }

    fri = fri * fr.cscale + fr.coffset;

    if (fr.fractal_flags & FRF_SIMPLE) {
        mfact = 0;
    } else {
        mfact = fri - floor(fri);
    }

    c0 = (int)floor(fri) % ncol;
    c1 = (c0 + 1) % ncol;

    c0 *= 4; c1 *= 4;

    #define MIX(a, b, F) ((b) * (F) + (a) * (1 - (F)))

    output[ri + 0] = (int)floor(MIX(col[c0 + 0], col[c1 + 0], mfact));
    output[ri + 1] = (int)floor(MIX(col[c0 + 1], col[c1 + 1], mfact));
    output[ri + 2] = (int)floor(MIX(col[c0 + 2], col[c1 + 2], mfact));
    output[ri + 3] = (int)floor(MIX(col[c0 + 3], col[c1 + 3], mfact));

}

void calc_cuda_init(fr_t fr, fr_col_t col) {
    if (!cuda_has_init) {
        int gpu_err_start = 0;
        gpuErrchk(hipMalloc((void **)&_gpu_err, sizeof(int)));
        gpuErrchk(hipMemcpy(_gpu_err, &gpu_err_start, sizeof(int), hipMemcpyHostToDevice));

        gpuErrchk(hipMalloc((void **)&_gpu_output, 4 * fr.w * fr.h));
        
        colnum = col.num;
        gpuErrchk(hipMalloc((void **)&_gpu_col, 4 * colnum));
        gpuErrchk(hipMemcpy(_gpu_col, col.col, 4 * colnum, hipMemcpyHostToDevice));
        cuda_has_init = true;
    }
}


// returns the grid from a block value such that b * grid_from_block(a, b) >= a,
// and grid_from_block(a, b) % b == 0
int grid_from_block(int a, int b) {
    return a / b + (a % b != 0);
}

void calc_cuda(fr_t fr, fr_col_t col, int tid, int threads, unsigned char * output) {
    calc_cuda_init(fr, col);

    dim3 dimBlock(16, 12);
    dim3 dimGrid(grid_from_block(fr.w,  dimBlock.x), 
                 grid_from_block(fr.h / threads, dimBlock.y));


    log_debug("cuda kernel launched at center: %.20lf,%.20lf, zoom: %lf, iter: %d with grid: (%d,%d), block (%d,%d)", fr.cX, fr.cY, fr.Z, fr.max_iter, dimGrid.x, dimGrid.y, dimBlock.x, dimBlock.y);

    // we dont need hipMalloc(), because of ZEROcopy buffers that the GPU and CPU can share sys memory
    cuda_kernel<<<dimGrid, dimBlock>>>(fr, tid, threads, _gpu_col, colnum, _gpu_output, _gpu_err);


    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());

    int res_err = 0;

    gpuErrchk(hipMemcpy(output, _gpu_output, 4 * fr.w * (fr.h / threads), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(&res_err, _gpu_err, sizeof(int), hipMemcpyDeviceToHost));

    if (res_err != 0) {
        log_error("result from cuda kernel is non-zero: %d\n", res_err);
    }

}

}

#undef cabs
#undef cabs2
#undef creal
#undef cimag




